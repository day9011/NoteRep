#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <cfloat>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "my-cuda-function-kernel-ansi.h"
#include <algorithm>
#include <iostream>
#include <fstream>
#include <limits>
#include <hipblas.h>

#define PI 3.1415926535897932384626433832795

#define CUDA_CALL(ret) \
{\
  if((ret) != hipSuccess) { \
  printf("Error at %s:%d\n", __FILE__, __LINE__); \
  printf("Error code %s\n", hipGetErrorString(ret)); \
  exit(-1); \
  } \
  hipDeviceSynchronize(); \
}

#define CURAND_CALL(x)\
{\
	if((x) != HIPRAND_STATUS_SUCCESS) {\
		printf("Error at %s:%d\n",__FILE__,__LINE__); \
		exit(-1); \
	}\
} 
using namespace std;

//compute matrix sum of one column
template<typename Real>
__device__
static Real _sum_col(const Real *x, MatrixDim d, int32_cuda col)
{
	Real sum = 0;
	for(int32_cuda i = 0; i < d.rows; i++)
		sum += x[i * d.stride + col];
	return sum;
}

//compute dot product between two vectors.
template<typename Real>
__device__
static void _VecVec(const Real *x, const Real *y, int32_cuda dim, Real *res)
{
	Real result = 0;
	for (int32_cuda i = 0; i < dim; i++)
		result += x[i] * y[i];
	*res = result;
}

//Scale all element
template<typename Real>
__device__
static void __Scale(Real *x, int32_cuda dim, Real alpha)
{
	for (int32_cuda i = 0; i < dim; i++)
		x[i] = x[i] * alpha;
}

//select max number which is not zero
template<typename Real>
__device__
static Real __Max(const Real *data, int32_cuda dim)
{
	Real ans = (Real)(-1000000);
	for (int32_cuda i = 0; i < dim; i++)
		if (data[i] > ans && data[i] != 0) ans = data[i];
	return ans;
}

//select max number index which value is not zero
template<typename Real>
__device__
static int32_cuda __Max_index(const Real *data, int32_cuda dim)
{
	Real ans = (Real)(-1000000);
	int32_cuda index = 0;
	for (int32_cuda i = 0; i < dim; i++)
		if (data[i] > ans && data[i] != 0) { ans = data[i]; index = i; }
	return index;
}


template<typename Real>
__device__
static Real __Sum(Real *data, int32_cuda dim)
{
	Real sum = 0;
	for (int32_cuda i = 0; i < dim; i++)
		sum += data[i];
	return sum;
}

template<typename Real>
__device__
static void __insert_sort(Real *__first, Real *__last)
{
	if (__first == __last)
		return;
	Real *p;
	for (Real *iter = __first + 1; iter != __last; ++iter)
	{
		Real tmp = *iter;
		for (p = iter; p != __first && tmp < *(p - 1); --p)
			*p = *(p - 1);
		*p = tmp;
	}
}

template<typename Real>
__device__
static Real* __partition(Real *__first, Real *__last, Real __pivot)
{
	while(true)
	{
		while (*__first < __pivot)
			++__first;
		--__last;
		while (__pivot < *__last)
			--__last;
		if(!(__first < __last))
			return __first;
		//swap two number, use plus function to swap two number will lost precision.
		{
			Real temp = *__first;
			*__first = *__last;
			*__last = temp;
		}
		++__first;
	}
}

template<typename Real>
__device__
static void _partition(Real *__first, Real *__nth, Real *__last)
{
	while(__last - __first > 3)
	{
		Real *__cut = __partition(__first, __last, *(__first + (__last - __first) / 2));
		if (__cut <= __nth)
			__first = __cut;
		else
			__last = __cut;
	}
	__insert_sort(__first, __last);
}

template<typename Real>
__global__
static void _gmm_select(const Real *data, MatrixDim d, Real *copydata, MatrixDim c_d, int32_cuda num_gselect, int32_cuda *gmm_selected)
{
	int32_cuda row = blockDim.x * blockIdx.x + threadIdx.x;
	if (row < d.rows)
	{
		_partition(copydata + row * c_d.stride, copydata + row * c_d.stride + c_d.cols - num_gselect, copydata + row * c_d.stride + c_d.cols);
		Real thresh = copydata[row * c_d.stride + c_d.cols - num_gselect];
		int32_cuda index = 0;
		for (int32_cuda j = 0; j < d.cols; j++)
			if (*(data + row * d.stride + j) >= thresh)
			{
				if(index < 20)
				{
					*(gmm_selected + row * num_gselect + index) = j;
					++index;
				}
				else break;
			}
		__syncthreads();
	}
}

template<typename Real>
__global__
static void _MatApplySoftMax(Real *data, MatrixDim d)
{
	int32_cuda i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < d.rows)
	{
		Real sum = 0;
		Real max = __Max(data + i * d.stride, d.cols);
		for (int32_cuda j = 0; j < d.cols; j++)
		{
			if (data[i * d.stride + j] != 0)
			{
				data[i * d.stride + j] = exp(data[i * d.stride + j] - max);
				sum += data[i * d.stride + j];
			}
		}
		__Scale(data + i * d.stride, d.cols, (Real)(1.0 / sum));
		__syncthreads();
	}
}

template<typename Real>
__global__
static void _compute_posterior(Real *loglikes, MatrixDim d_log, Real min_post)
{
	int32_cuda i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < d_log.rows)
	{
		if (min_post != 0.0)
		{
			int32_cuda max_index = __Max_index(loglikes + i * d_log.stride, d_log.cols);
			for (int32_cuda j = 0; j < d_log.cols; j++)
				if (loglikes[i * d_log.stride + j] < min_post)
					loglikes[i * d_log.stride + j] = 0.0;
			Real sum = __Sum(loglikes + i * d_log.stride, d_log.cols);
			if (sum == 0.0)
				loglikes[i * d_log.stride + max_index] = 1.0;
			else
				__Scale(loglikes + i * d_log.stride, d_log.cols, (Real)(1.0 / sum));
		}
		__syncthreads();
	}
}

//get a vector by sum of elements of every cols
template<typename Real>
__global__
static void _add_cols_mat_to_vec(const Real *mat, MatrixDim d, Real *vec)
{
	int32_cuda i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < d.cols)
	{
		vec[i] = _sum_col(mat, d, i);
	}
}

template<typename Real>
__global__
static void _scale_linear(Real *A, int32_cuda dim, Real alpha)
{
	int32_cuda i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < dim)
	{
		A[i] = A[i] * alpha;
	}
}

template<typename Real>
__global__
static void _scale_diag_numsp(Real *A, int32_cuda nums, MatrixDim d, Real alpha)
{
	int32_cuda i = blockIdx.x * blockDim.x + threadIdx.x;
	int32_cuda j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < nums)
	{
		if (j < d.cols)
		{
			int32_cuda index = (j + 1) * (j + 2) / 2 - 1;
 			A[i * d.stride + index] = A[i * d.stride + index] * alpha;
			__syncthreads();
		}
	}
}

template <typename Real>
__global__
static void _my_addvec2(Real *A, const Real *x, int32_cuda dim, Real alpha)
{
	int32_cuda i = blockIdx.x * blockDim.x + threadIdx.x;
	int32_cuda j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i <= j && j < dim)
	{
		A[j * (j + 1) / 2 + i] = x[j] * x[i] * alpha;
		__syncthreads();
	}
}

template<typename Real>
__global__
static void _my_addvec3(Real *A, int32_cuda numA, const Real *x, MatrixDim d, Real alpha)
{
    int32_cuda i = blockIdx.x * blockDim.x + threadIdx.x;
    int32_cuda j = blockIdx.y * blockDim.y + threadIdx.y;
    int32_cuda k = blockIdx.z * blockDim.z + threadIdx.z;
	/* int maxi = 0, maxj = 0, maxk = 0; */
	/* if (i > maxi) maxi = i; */
	/* if (j > maxj) maxj = j; */
	/* if (k > maxk) maxk = k; */
    int32_cuda stride = d.cols * (d.cols + 1) / 2;
    if (i < numA && j < d.cols && k < d.cols)
    {
	    if(k <= j)
	    {
			A[i * stride + j * (j + 1) / 2 + k] = x[i * d.stride + j] * x[i * d.stride + k] * alpha;
		}
	    __syncthreads();
		/* if (i > (numA - 3) && j > (d.cols - 3) && k > (d.cols - 3)) */
		/*     printf("\nmax i: %d, max j: %d, max k: %d\n", maxi, maxj, maxk); */
	}
}

template<typename Real>
__global__
static void printdata(Real *data, MatrixDim d)
{
	for (int r = 0; r < 2; r++)
	{
		printf("[");
		for (int c = 0; c < d.cols; c++)
			printf(" %g", data[1000 * d.stride + c]);
		printf(" ]");
	}
}

template<typename Real>
__host__
static void _my_cuda_AddMatColsToVec(const Real *mat, MatrixDim d, Real *vec, int32_cuda Gr, int32_cuda Bl)
{
	_add_cols_mat_to_vec<<<Gr, Bl>>>(mat, d, vec);
}

template<typename Real>
__host__
static void _my_cuda_MatApplySoftMax(Real *data, MatrixDim d, int32_cuda Gr, int32_cuda Bl)
{
	_MatApplySoftMax<<<Gr, Bl>>>(data, d);
}

template<typename Real>
__host__
static void _my_cuda_scale_diag_numsp(Real *A, int32_cuda nums, MatrixDim d, Real alpha, dim3 Gr, dim3 Bl)
{
	_scale_diag_numsp<<<Gr,Bl>>>(A, nums, d, alpha);
}

template<typename Real>
__host__
static void _my_cuda_scale_linear(int32_cuda Gr, int32_cuda Bl, Real *A, int32_cuda dim, Real alpha)
{
	_scale_linear<<<Gr, Bl>>>(A, dim, alpha);
}

template<typename Real>
__host__
static void _my_cuda_gmm_select(int32_cuda Gr, int32_cuda Bl, const Real *data, MatrixDim d, int32_cuda num_gselect, int32_cuda *gmm_out)
{
	Real *copydata;
	size_t pitch;
	MatrixDim c_d;
	c_d.rows = d.rows;
	c_d.cols = d.cols;
	CUDA_CALL(hipMallocPitch((void **)&copydata, &pitch, d.cols * sizeof(Real), d.rows));
	c_d.stride = pitch / sizeof(Real);
	CUDA_CALL(hipMemcpy2D(copydata, c_d.stride * sizeof(Real), data, d.stride * sizeof(Real), d.cols * sizeof(Real), d.rows, hipMemcpyDeviceToDevice));
	hipDeviceSynchronize();
	_gmm_select<<<Gr, Bl>>>(data, d, copydata, c_d, num_gselect, gmm_out);
	CUDA_CALL(hipFree(copydata));
}


template<typename Real>
__host__
static void _my_cuda_compute_fft(Real *data, int32_cuda dim)
{
	hipfftComplex *CompData = (hipfftComplex *)malloc(dim * sizeof(hipfftComplex));
	for (int32_cuda i = 0; i < dim; i++)
	{
		CompData[i].x = data[i];
		CompData[i].y = 0;
	}
	hipfftComplex *devData;
	CUDA_CALL(hipMalloc((void **)&devData, dim * sizeof(hipfftComplex)));
	CUDA_CALL(hipMemcpy(devData, CompData, dim * sizeof(CompData), hipMemcpyHostToDevice));

	hipfftHandle plan;
	hipfftPlan1d(&plan, dim, HIPFFT_C2C, 1);
	hipfftExecC2C(plan, devData, devData, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	CUDA_CALL(hipMemcpy(CompData, devData, dim * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
	for (int32_cuda i = 0; i < dim / 2; i++)
	{
		data[2 * i] = CompData[i].x;
		data[2 * i + 1] = CompData[i].y;
	}
	data[1] = CompData[dim / 2].x;
	CUDA_CALL(hipFree(devData));
	free(CompData);
}

// 基于cuBlas实现的矩阵和向量的乘法
template<typename Real>
__host__
void _add_mat_vec(const Real *gamma, const Real *Sigma_inv_M_, Real *data, int32_cuda gamma_size, int32_cuda Sigma_row, int32_cuda Sigma_col){
	Real alpha = 1.0f;
	Real beta = 0.0f;
	hipblasHandle_t handle;  
	
	hipblasCreate(&handle); 	
	mySgemv(handle, HIPBLAS_OP_N, Sigma_col, Sigma_row, &alpha, Sigma_inv_M_, Sigma_col, gamma, 1, &beta, data, 1);
	hipblasDestroy(handle);	
}

hipblasStatus_t mySgemv(hipblasHandle_t &handle, hipblasOperation_t trans, int m, int n, const float *alpha, const float *A, int lda, const float *x, int incx, const float *beta, float *y, int incy){
	return hipblasSgemv(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy);
}

hipblasStatus_t mySgemv(hipblasHandle_t &handle, hipblasOperation_t trans, int m, int n, const double *alpha, const double *A, int lda, const double *x, int incx, const double *beta, double *y, int incy){
	return hipblasDgemv(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy);

}

//Parallel memory copy to cuda variables
template<typename Real>
__global__
static void _parallel_memcpy(Real *des, Real *src, int32_cuda sizes, int32_cuda rows, int32_cuda cols, int32_cuda stride){
	if(threadIdx.x <= cols && threadIdx.x <= stride){
		int32_cuda des_id = threadIdx.x + blockIdx.x * cols;
		int32_cuda src_id = threadIdx.x + blockIdx.x * stride;
		int32_cuda gen_stride = gridDim.x;
    	int32_cuda stridex = 0;

		for(;src_id + stridex * stride < sizes * rows * stride; stridex += gen_stride){	
			*(des + des_id + stridex * cols) = *(src + src_id + stridex * stride);
		}
	}
}

// compute vecvec between two matrixs
template<typename Real>
__global__
static void _MatVecVec(const Real *A, const Real *B, int32_cuda rows, int32_cuda cols, int32_cuda strideA, int32_cuda strideB, Real *out)
{
	int32_cuda i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < rows)
	{
		_VecVec(A + i * strideA, B + i * strideB, cols, out + i);
	}
}

template<typename Real>
__global__
static void _LogLikelihoodsPreselect(const int32_cuda *gselect, int32_cuda gselect_rows, int32_cuda gselect_cols, const Real *features, MatrixDim d_features, const Real *gconsts_, int32_cuda dim_gconsts, const Real *means_invcovars_, MatrixDim d_means, const Real *data_sqs, const Real *inv_covars_, int32_cuda spdim, Real *loglikes, MatrixDim d_loglikes)
{
	int32_cuda i = blockIdx.x * blockDim.x + threadIdx.x;
	int32_cuda j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < d_features.rows && j < gselect_cols)
	{
		int32_cuda idx = gselect[i * gselect_cols + j];
		Real means_inv_data = 0, data_sq_inv = 0;
		_VecVec(means_invcovars_ + idx * d_means.stride, features + i * d_features.stride, d_features.cols, &means_inv_data);
		_VecVec(data_sqs + i * spdim, inv_covars_ + idx * spdim, spdim, &data_sq_inv);
		loglikes[i * d_loglikes.stride + idx] = gconsts_[idx] + means_inv_data - data_sq_inv;
		__syncthreads();
	}
}

template<typename Real>
__host__
static void _my_cuda_LogLikelihoodsPreselect(const int32_cuda *gselect, int32_cuda gselect_rows, int32_cuda gselect_cols, const Real *features, MatrixDim d_features, const Real *gconsts_, int32_cuda dim_gconsts, const Real *means_invcovars_, MatrixDim d_means, const Real *data_sqs, const Real *inv_covars_, int32_cuda spdim, Real *loglikes, MatrixDim d_loglikes, dim3 Gr, dim3 Bl)
{
	_LogLikelihoodsPreselect<<<Gr, Bl>>>(gselect, gselect_rows, gselect_cols, features, d_features, gconsts_, dim_gconsts, means_invcovars_, d_means, data_sqs, inv_covars_, spdim, loglikes, d_loglikes);
}


template<typename Real>
__host__
static void _my_cuda_MatVecVec(const Real *A, const Real *B, MatrixDim dA, MatrixDim dB, Real *out, int32_cuda Gr, int32_cuda Bl)
{
	_MatVecVec<<<Gr, Bl>>>(A, B, dA.rows, dA.cols, dA.stride, dB.stride, out);
}

template<typename Real>
__host__
static void _my_parallel_memcpy(Real *des, Real *src, int32_cuda sizes, int32_cuda rows, int32_cuda cols, int32_cuda stride, int32_cuda Gr, int32_cuda Bl){
	_parallel_memcpy<<<Gr ,Bl>>>(des, src, sizes, rows, cols, stride);
}

//Parallel memory copy from vec to mat 
//by default: (rows - 1) * cols <= element(src) <= rows * cols
template<typename Real>
__global__ 
static void _vec_memcpy(const Real *src, int32_cuda len, int32_cuda blank, Real *des, int32_cuda rows, int32_cuda cols, int32_cuda stride){
	if(threadIdx.x >= cols || blockIdx.x >= rows) return;
	int32_cuda row = blockIdx.x;
	int32_cuda row_stride = gridDim.x;
	
	while(row < rows){
		int32_cuda col = threadIdx.x;
		int32_cuda col_stride = blockDim.x;
		int32_cuda cur_idx = row * blank;
		while(col < cols && cur_idx + col + row*cols < len){
			*(des + col + row * stride) = *(src + cur_idx + col + row * cols);
			col += col_stride;
		}
		row += row_stride;
	}
}

template<typename Real>
__host__
static void _cuda_vec_memcpy(const Real *src, int32_cuda len, int32_cuda blank, Real *des, int32_cuda rows, int32_cuda cols, int32_cuda stride, int32_cuda Gr, int32_cuda Bl){
	_vec_memcpy<<<Gr ,Bl>>>(src, len, blank, des, rows, cols, stride);
}

// computing mel energies
// For efficiency: grid = frames, block = bins_len
template<typename Real>
__global__
static void _parallel_mel_vecvec(const Real *voice, const int32_cuda rows, const int32_cuda cols, const int32_cuda *offset, const int32_cuda *bins_len, const Real *bins, const int32_cuda bin_rows, const int32_cuda bin_cols, Real *mel_energies_out, const int32_cuda mel_stride, int32_cuda htk_mode){
	int32_cuda tid = threadIdx.x;
	int32_cuda frame_num = blockIdx.x;
	Real temp = 0.0;
	
	for(int32_cuda dimx = 0; dimx < *(bins_len + tid); dimx += 1){			
		temp += *(voice + frame_num * cols + (*(offset + tid)) + dimx) * (*(bins + dimx + tid * bin_cols));
	}
	*(mel_energies_out + tid + frame_num * mel_stride) = (htk_mode == 0 && temp < 1)? 1 : temp;
}

template<typename Real>
__host__
static void _my_cuda_parallel_mel_vecvec(const Real *voice, const int32_cuda rows, const int32_cuda cols, const int32_cuda *offset, const int32_cuda *bins_len, const Real *bins, const int32_cuda bin_rows, const int32_cuda bin_cols, Real *mel_energies_out, int32_cuda mel_stride, int32_cuda htk_mode, int32_cuda Gr){
	_parallel_mel_vecvec<<<Gr, bin_rows>>>(voice, rows, cols, offset, bins_len, bins, bin_rows, bin_cols, mel_energies_out, mel_stride, htk_mode);
}

//functions for extract window
__global__
static void _F_cuda_process_gauss_val(float *val, int32_cuda len){
	int32_cuda idx = threadIdx.x + blockIdx.x * blockDim.x;
	int32_cuda stride = gridDim.x * blockDim.x;
	while(idx < len){
		*(val + idx) = sqrtf(-2 * logf(*(val + idx))) * cospif(*(val + idx + len) * 2);
		idx += stride;
	}
}

__host__
static void _F_process_gauss_val(float *val, int32_cuda len, const int32_cuda Gr, const int32_cuda Bl){
	_F_cuda_process_gauss_val<<<Gr,Bl>>>(val,len);
}

__global__ 
static void _D_cuda_process_gauss_val(double *val, int32_cuda len){
	int32_cuda idx = threadIdx.x + blockIdx.x * blockDim.x;
	int32_cuda stride = gridDim.x * blockDim.x;
	while(idx < len){
		*(val + idx) = sqrt(-2 * log(*(val + idx))) * cospi(*(val + idx + len) * 2);
		idx += stride;
	}
}

__host__
static void _D_process_gauss_val(double *val, int32_cuda len, const int32_cuda Gr, const int32_cuda Bl){
	_D_cuda_process_gauss_val<<<Gr,Bl>>>(val, len);
}

//block = numframes, thread = cols
template<typename Real>
__global__
static void _cuda_dither(Real *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda matrix_stride, Real dither_value, float *gauss_val){
		
	int32_cuda row = blockIdx.x;
	int32_cuda col = threadIdx.x;
	if(row > rows || col > cols) return;
	while(row < rows){
		for(int32_cuda i = col; i < cols; i += blockDim.x)
			*(waveform + row * matrix_stride + i) += dither_value * (*(gauss_val + row * cols + i));
		row += gridDim.x;
	}
}

template<typename Real>
__host__
static void _my_cuda_dither(Real *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda matrix_stride, Real dither_value, const int32_cuda Gr, const int32_cuda Bl){
	hiprandGenerator_t gen;
	clock_t seed = clock();
	float *gauss_val;
	CUDA_CALL(hipMalloc((void **)&gauss_val, sizeof(float) * rows * cols));	
	CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
	CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, seed));
	CURAND_CALL(hiprandGenerateNormal(gen, gauss_val, rows * cols, 0, 1));
	_cuda_dither<<<Gr,Bl>>>(waveform, rows, cols, matrix_stride, dither_value, gauss_val);
	CURAND_CALL(hiprandDestroyGenerator(gen));
	CUDA_CALL(hipFree(gauss_val));
}

//block = numframes
template<typename Real>
__global__
static void _cuda_preemphasize(Real *waveform, int32_cuda rows, int32_cuda cols, int32_cuda matrix_stride, Real coeff){
	int32_cuda idx = blockIdx.x;
	if(coeff == 0.0 ||  idx >= rows) return;
	for (int32_cuda i = matrix_stride * idx + cols - 1; i > matrix_stride * idx; i--)
    	*(waveform + i) -= coeff * *(waveform + i - 1);
  	*(waveform + matrix_stride * idx) -= coeff * *(waveform + matrix_stride * idx);
}

template<typename Real>
__host__
static void _my_cuda_preemphasize(Real *waveform, int32_cuda rows, int32_cuda cols, int32_cuda matrix_stride, Real coeff, const int32_cuda Gr, const int32_cuda Bl){
	_cuda_preemphasize<<<Gr,Bl>>>(waveform, rows, cols, matrix_stride, coeff);
}

template<typename Real>
__global__
static void _cuda_wave_sum(const Real *waveform, int32_cuda rows, int32_cuda cols, int32_cuda matrix_stride, Real *wave_sum){
	int32_cuda row = blockIdx.x;
	int32_cuda stride = gridDim.x;
	if(row >= rows) return;
	while(row < rows){
		for(int32_cuda i = 0; i < cols; i++){
			*(wave_sum + row) = (*(wave_sum + row)) + (*(waveform + row * matrix_stride + i));
		}
		*(wave_sum + row) = (*(wave_sum + row)) / cols;
		row += stride;
	}
}

template<typename Real>
__host__
static void _my_cuda_wave_sum(const Real *waveform, int32_cuda rows, int32_cuda cols, int32_cuda matrix_stride, Real *wave_sum, const int32_cuda Gr, const int32_cuda Bl){
	_cuda_wave_sum<<<Gr,Bl>>>(waveform, rows, cols, matrix_stride, wave_sum);
	
}

//for wave mat elementwise multiple a window vector, same col length
template<typename Real>
__global__
static void _cuda_wave_mul(Real *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda stride, const Real *window){
	int row = blockIdx.x;
	for(int32_cuda col = threadIdx.x; col < cols; col += blockDim.x)
		*(waveform + row * stride + col) *= *(window + col);
}

template<typename Real>
__host__
static void _my_cuda_wave_mul(Real *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda stride, const Real *window, const int32_cuda Gr, const int32_cuda Bl){
	_cuda_wave_mul<<<Gr, Bl>>>(waveform, rows, cols, stride, window);
}


//seting the elements between cols and matrix_stride 0
//block = numframes, thread = elements
template<typename Real>
__global__
static void _cuda_set_zero(Real *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda matrix_stride){
	int32_cuda col = threadIdx.x;
	int32_cuda col_stride = blockDim.x;
	int32_cuda row = blockIdx.x;
	int32_cuda row_stride = gridDim.x;
	if(row >= rows || col >= matrix_stride - cols) return;
	while (row < rows){
		for(int32_cuda i = col; i < matrix_stride - cols; i += col_stride)
			*(waveform + i + cols + row * matrix_stride) = 0; 
		row += row_stride;
	}	
}

template<typename Real>
__host__
static void _my_cuda_set_zero(Real *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda matrix_stride, const int32_cuda Gr, const int32_cuda Bl){
	_cuda_set_zero<<<Gr,Bl>>>(waveform, rows, cols, matrix_stride);
}

//block = numframes, thread = element
template<typename Real>
__global__
static void _cuda_wave_dc_offset(Real *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda matrix_stride, const Real *wave_sum){
	int32_cuda row = blockIdx.x;
	int32_cuda row_stride = gridDim.x;
	int32_cuda col = threadIdx.x;
	int32_cuda col_stride = blockDim.x;
	if(row >= rows || col >= cols) return;
	while(row < rows){
		for(int32_cuda i = col; i < cols; i += col_stride)
			*(waveform + row * matrix_stride + i) = (*(waveform + row * matrix_stride + i)) - (*(wave_sum + row)); 
		row += row_stride;
	}

}

template<typename Real>
__host__
static void _my_cuda_wave_dc_offset(Real *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda matrix_stride, const Real *wave_sum, const int32_cuda Gr, const int32_cuda Bl){
	 _cuda_wave_dc_offset<<<Gr,Bl>>>(waveform, rows, cols, matrix_stride, wave_sum);
}

// For parallel computing log energy function
template<typename Real>
__global__ 
static void _get_element(Real *des, const Real *src, const int32_cuda rows, const int32_cuda cols, int32_cuda stride, int32_cuda bidx, int32_cuda acc, const Real lower_bound){
	if (blockIdx.x >= rows) return;
	int32_cuda tid = blockIdx.x;
	*(des + tid + bidx) = *(src + tid * stride + bidx + tid * acc) < lower_bound ? lower_bound : *(src + tid * stride + bidx + tid * acc);
}

__global__
static void _F_log(float *des){
	*(des + blockIdx.x) = logf(*(des + blockIdx.x));
}

__global__
static void _D_log(double *des){
	*(des + blockIdx.x) = log(*(des + blockIdx.x));
}

__host__
static void _F_log_energy(const float *src, int32_cuda rows, int32_cuda cols, int32_cuda stride, float *des){
	float alpha = 1.0f;
	float beta = 0.0f;
	float *tmp;
    CUDA_CALL(hipMalloc((void **)&tmp, sizeof(float) * rows * rows));
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rows, rows, stride, &alpha, src, stride, src, stride, &beta, tmp, rows);
	_get_element<<<rows,1>>>(des, tmp, rows, rows, rows, 0, 1, numeric_limits<float>::min());
	_F_log<<<rows,1>>>(des);

	CUDA_CALL(hipFree(tmp));
}

__host__
static void _D_log_energy(const double *src, int32_cuda rows, int32_cuda cols, int32_cuda stride, double *des){
	double alpha = 1.0f;
	double beta = 0.0f;
	double *tmp;
    CUDA_CALL(hipMalloc((void **)&tmp, sizeof(double) * rows * stride));	
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rows, rows, cols, &alpha, src, cols, src, cols, &beta, tmp, rows);
	hipblasDestroy(handle);
	_get_element<<<rows,1>>>(des, tmp, rows, rows, stride, 0, 1, numeric_limits<double>::min());
	_D_log<<<rows,1>>>(des);
	CUDA_CALL(hipFree(tmp));
}

// functions for srfft
template<typename Real>
__device__
static void _swap(Real &a, Real &b){
	Real temp;
	temp = a;
	a = b;
	b = temp;
}

template<typename Real>
__device__
static void bitrp (Real *xreal, Real *ximag, int32_cuda n)
{
    // 位反转置换 Bit-reversal Permutation
	int32_cuda i, j, a, b, p;
	for (i = 1, p = 0; i < n; i *= 2)
	    p ++;
	for (i = 0; i < n; i ++){	
		a = i;
		b = 0;	
		for (j = 0; j < p; j ++){
			b = (b << 1) + (a & 1);    // b = b * 2 + a % 2;
			a >>= 1;        // a = a / 2;
		}
		if ( b > i){
			_swap (*(xreal + i), *(xreal + b));
			_swap (*(ximag + i), *(ximag + b));
		}
	}
}

template<typename Real>
__device__
static void FFT(Real *xreal, Real *ximag, Real *wreal, Real *wimag, const int32_cuda n)
{
    // 快速傅立叶变换，将复数 x 变换后仍保存在 x 中，xreal, ximag 分别是 x 的实部和虚部
	Real treal, timag, ureal, uimag;
	int32_cuda m, k, j, t, index1, index2;
	bitrp (xreal, ximag, n);
	for (m = 2; m <= n; m *= 2){
		for (k = 0; k < n; k += m){
			for (j = 0; j < m / 2; j ++){
				index1 = k + j;	
				index2 = index1 + m / 2;
				t = n * j / m;    // 旋转因子 w 的实部在 wreal [] 中的下标为 t
				treal = *(wreal + t) * *(xreal + index2) - *(wimag + t) * *(ximag + index2);
				timag = *(wreal + t) * *(ximag + index2) + *(wimag + t) * *(xreal + index2);
				ureal = *(xreal + index1);
				uimag = *(ximag + index1);
				*(xreal + index1) = ureal + treal;
				*(ximag + index1) = uimag + timag;	
				*(xreal + index2) = ureal - treal;
				*(ximag + index2) = uimag - timag;
			}
		}
	}
}

template<typename Real>
__device__
static void _cuda_memcpy(Real *des, Real *src, int32_cuda n){
	for(int32_cuda i = 0; i < n; i++)
		*(des + i) = *(src + i);
}

template<typename Real>
__device__
inline void _complexAddProduct(const Real &a_re, const Real &a_im, const Real &b_re, const Real &b_im, Real *c_re, Real *c_im ){
	*c_re += b_re*a_re - b_im*a_im;
	*c_im += b_re*a_im + b_im*a_re;
}

template<typename Real>
__device__
inline void _complexMul(const Real &a_re, const Real &a_im, Real *b_re, Real *b_im){
		Real tmp_re = (*b_re * a_re) - (*b_im * a_im);
	   *b_im = *b_re * a_im + (*b_im * a_re);
	   *b_re = tmp_re;
}

template<typename Real>
__device__
inline void _complexImExp(Real x, Real *a_re, Real *a_im){
	*a_re = cos(x);
	*a_im = sin(x);
}

template<typename Real>
__device__
static void _FFT_trans(Real *data, int32_cuda stride){
	int32_cuda N = stride, N2 = N/2;
  	Real rootN_re, rootN_im;  // exp(-2pi/N), forward; exp(2pi/N), backward
  int32_cuda forward_sign = -1;
  _complexImExp(static_cast<Real>(2.0 * PI/N *forward_sign), &rootN_re, &rootN_im);
  Real kN_re = -forward_sign, kN_im = 0.0;  // exp(-2pik/N), forward; exp(-2pik/N), backward
  // kN starts out as 1.0 for forward algorithm but -1.0 for backward.
  for (int32_cuda k = 1; 2*k <= N2; k++) {
    _complexMul(rootN_re, rootN_im, &kN_re, &kN_im);

    Real Ck_re, Ck_im, Dk_re, Dk_im;
    // C_k = 1/2 (B_k + B_{N/2 - k}^*) :
    Ck_re = 0.5 * (data[2*k] + data[N - 2*k]);
    Ck_im = 0.5 * (data[2*k + 1] - data[N - 2*k + 1]);
    // re(D_k)= 1/2 (im(B_k) + im(B_{N/2-k})):
    Dk_re = 0.5 * (data[2*k + 1] + data[N - 2*k + 1]);
    // im(D_k) = -1/2 (re(B_k) - re(B_{N/2-k}))
    Dk_im =-0.5 * (data[2*k] - data[N - 2*k]);
    // A_k = C_k + 1^(k/N) D_k:
    data[2*k] = Ck_re;  // A_k <-- C_k
    data[2*k+1] = Ck_im;
    // now A_k += D_k 1^(k/N)
    _complexAddProduct(Dk_re, Dk_im, kN_re, kN_im, &(data[2*k]), &(data[2*k+1]));

    int32_cuda kdash = N2 - k;
    if (kdash != k) {
      // Next we handle the index k' = N/2 - k.  This is necessary
      // to do now, to avoid invalidating data that we will later need.
      // The quantities C_{k'} and D_{k'} are just the conjugates of C_k
      // and D_k, so the equations are simple modifications of the above,
      // replacing Ck_im and Dk_im with their negatives.
      data[2*kdash] = Ck_re;  // A_k' <-- C_k'
      data[2*kdash+1] = -Ck_im;
      // now A_k' += D_k' 1^(k'/N)
      // We use 1^(k'/N) = 1^((N/2 - k) / N) = 1^(1/2) 1^(-k/N) = -1 * (1^(k/N))^*
      // so it's the same as 1^(k/N) but with the real part negated.
      _complexAddProduct(Dk_re, -Dk_im, -kN_re, kN_im, &(data[2*kdash]), &(data[2*kdash+1]));
    }
  }

  {  // Now handle k = 0.
    // In simple terms: after the complex fft, data[0] becomes the sum of real
    // parts input[0], input[2]... and data[1] becomes the sum of imaginary
    // pats input[1], input[3]...
    // "zeroth" [A_0] is just the sum of input[0]+input[1]+input[2]..
    // and "n2th" [A_{N/2}] is input[0]-input[1]+input[2]... .
    Real zeroth = data[0] + data[1],
        n2th = data[0] - data[1];
    data[0] = zeroth;
    data[1] = n2th; 
  }
}

//global function for fft
//1. change array form (first part: real, second part: image)
//2. perform fft
//3. change back
template<typename Real>
__global__
static void _cuda_srfft(Real *wave, int32_cuda rows, int32_cuda stride, Real *temp_buffer, Real *wreal, Real *wimag){
	if(blockIdx.x >= rows) return;
	int32_cuda N = stride / 2;
	int32_cuda tid = blockIdx.x;
	for(int32_cuda i = 0; i < N; i++){
		*(wave + tid * stride + i) = *(wave + tid * stride + i * 2);
		*(temp_buffer + tid * stride + i) = *(wave + tid * stride + i * 2 + 1);
	}
	_cuda_memcpy(wave + tid * stride + N, temp_buffer + tid * stride, N);
	FFT(wave + tid * stride, wave + tid * stride + N, wreal, wimag, N);
	_cuda_memcpy(temp_buffer + tid * stride, wave + tid * stride + N, N);

	for(int32_cuda i = N - 1; i > 0; i--){
		*(wave + tid * stride + i * 2) = *(wave + tid * stride + i);
		*(wave + tid * stride + i * 2 + 1) = *(temp_buffer + tid * stride + i);
	}
	*(wave + tid * stride + 1) = *(temp_buffer + tid * stride);
	_FFT_trans(wave + tid * stride, stride);
}

template<typename Real>
__host__
static void _my_cuda_srfft(Real *wave, int32_cuda rows, int32_cuda stride, Real *temp_buffer){
	Real *wreal, *wimag, *dev_wreal, *dev_wimag, arg, treal, timag;
	int32_cuda n = stride / 2;
	wreal = (Real *)malloc(sizeof(Real) * n);
	wimag = (Real *)malloc(sizeof(Real) * n);
	arg = - 2.0 * PI / n;
	treal = cos (arg);
	timag = sin (arg);
	*wreal = 1.0;
	*wimag = 0.0;
	
	for (int32_cuda j = 1; j < n / 2; j ++)
	{
		*(wreal + j) = *(wreal + j - 1) * treal - *(wimag + j - 1) * timag;
		*(wimag + j) = *(wreal + j - 1) * timag + *(wimag + j - 1) * treal;
	}
	CUDA_CALL(hipMalloc((void **)&dev_wreal, sizeof(Real) * n));
	CUDA_CALL(hipMalloc((void **)&dev_wimag, sizeof(Real) * n));
	CUDA_CALL(hipMemcpy(dev_wreal, wreal, sizeof(Real) * n, hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_wimag, wimag, sizeof(Real) * n, hipMemcpyHostToDevice));
	_cuda_srfft<<<rows, 1>>>(wave, rows, stride, temp_buffer, dev_wreal, dev_wimag);
	

	free(wreal);
	free(wimag);
	CUDA_CALL(hipFree(dev_wreal));
	CUDA_CALL(hipFree(dev_wimag));
}

//compute power spectrum of FFT
template<typename Real>
__global__
static void _compute_power(Real *waveform, int32_cuda rows, int32_cuda cols, int32_cuda stride){
	int32_cuda half_dim = stride / 2;
	int32_cuda row = blockIdx.x; 
	Real first_energy = *(waveform + row * stride) * *(waveform + row * stride),
      last_energy = *(waveform + row * stride + 1) * *(waveform + row * stride + 1);  // handle this special case
  for (int32_cuda i = 1; i < half_dim; i++) {
    Real real = *(waveform + row * stride + i * 2), im = *(waveform + row * stride + i * 2 + 1);
    *(waveform + row * stride + i) = real * real + im * im;
  }
  *(waveform + row * stride) = first_energy;
  *(waveform + row * stride + half_dim) = last_energy;
}
	
template<typename Real>
__host__
static void _my_cuda_compute_power(Real *waveform, int32_cuda rows, int32_cuda cols, int32_cuda stride){
	_compute_power<<<rows, 1>>>(waveform, rows, cols, stride);
}

//set 0-th energy 
template<typename Real>
__global__
static void _set_energy(Real *des, const int32_cuda stride, const Real energy_floor, const Real log_energy_floor, const Real *src){
	int32_cuda tid = blockIdx.x;
	Real log_energy = *(src + tid);
	if(energy_floor > 0.0 && log_energy < log_energy_floor)
		log_energy = log_energy_floor;
	*(des + tid * stride) = log_energy;
}

template<typename Real>
__host__
static void _my_cuda_set_energy(Real *des, const int32_cuda rows, const int32_cuda stride, const Real energy_floor, const Real log_energy_floor, const Real *src){
	_set_energy<<<rows, 1>>>(des, stride, energy_floor, log_energy_floor, src);
}

template <typename Real>
__host__
static void _my_cuda_addvec2(dim3 Gr, dim3 Bl, Real *A, const Real *x, int32_cuda dim, Real alpha)
{
	_my_addvec2<<<Gr, Bl>>>(A, x, dim, alpha);
}

template <typename Real>
__host__
static void _my_cuda_addvec3(dim3 Gr, dim3 Bl, Real *A, int32_cuda numA, const Real *x, MatrixDim d, Real alpha)
{
	_my_addvec3<<<Gr, Bl>>>(A, numA, x, d, alpha);
}

template <typename Real>
__host__
static void _my_cuda_compute_posterior(int32_cuda Gr, int32_cuda Bl, Real *loglikes, MatrixDim d_log, Real min_post)
{
	_compute_posterior<<<Gr, Bl>>>(loglikes, d_log, min_post);
}

void _F_my_cuda_compute_posterior(float *loglikes, MatrixDim d_log, float min_post, int32_cuda Gr, int32_cuda Bl)
{
	_my_cuda_compute_posterior(Gr, Bl, loglikes, d_log, min_post);
}

void _D_my_cuda_compute_posterior(double *loglikes, MatrixDim d_log, double min_post, int32_cuda Gr, int32_cuda Bl)
{
	_my_cuda_compute_posterior(Gr, Bl, loglikes, d_log, min_post);
}

//define in my-cuda-function-kernel-ansi.h
void _F_my_cuda_compute_fft(float *data, int32_cuda dim)
{
	_my_cuda_compute_fft(data, dim);
}

void _D_my_cuda_compute_fft(double *data, int32_cuda dim)
{
	_my_cuda_compute_fft(data, dim);
}

void _F_my_cuda_gmm_select(int32_cuda Gr, int32_cuda Bl, const float *data, MatrixDim d, int32_cuda num_gselect, int32_cuda *gmm_out)
{
	_my_cuda_gmm_select(Gr, Bl, data, d, num_gselect, gmm_out);
}

void _D_my_cuda_gmm_select(int32_cuda Gr, int32_cuda Bl, const double *data, MatrixDim d, int32_cuda num_gselect, int32_cuda *gmm_out)
{
	_my_cuda_gmm_select(Gr, Bl, data, d, num_gselect, gmm_out);
}

void _F_my_cuda_add_mat_vec(const float *gamma, const float *Sigma_inv_M_, float *data, int32_cuda gamma_size, int32_cuda Sigma_row, int32_cuda Sigma_col){
    _add_mat_vec(gamma, Sigma_inv_M_, data, gamma_size, Sigma_row, Sigma_col);
}

void _D_my_cuda_add_mat_vec(const double *gamma, const double *Sigma_inv_M_, double *data, int32_cuda gamma_size, int32_cuda Sigma_row, int32_cuda Sigma_col){
    _add_mat_vec(gamma, Sigma_inv_M_, data, gamma_size, Sigma_row, Sigma_col);
}

void _F_my_cuda_parallel_memcpy(float *des, float *src, int32_cuda sizes, int32_cuda rows, int32_cuda cols, int32_cuda stride, int32_cuda Gr, int32_cuda Bl){
	_my_parallel_memcpy(des, src, sizes, rows, cols, stride, Gr, Bl);	
}

void _D_my_cuda_parallel_memcpy(double *des, double *src, int32_cuda sizes, int32_cuda rows, int32_cuda cols, int32_cuda stride, int32_cuda Gr, int32_cuda Bl){
	_my_parallel_memcpy(des, src, sizes, rows, cols, stride, Gr, Bl);
}

void _F_my_cuda_vec_memcpy(const float *src, int32_cuda len, int32_cuda blank, float *des, int32_cuda rows, int32_cuda cols, int32_cuda stride, int32_cuda Gr, int32_cuda Bl){
	_cuda_vec_memcpy(src, len, blank, des, rows, cols, stride, Gr, Bl);
}

void _D_my_cuda_vec_memcpy(const double *src, int32_cuda len, int32_cuda blank, double *des, int32_cuda rows, int32_cuda cols, int32_cuda stride, int32_cuda Gr, int32_cuda Bl){
	_cuda_vec_memcpy(src, len, blank, des, rows, cols, stride, Gr, Bl);
}

void _F_my_cuda_parallel_mel_vecvec(const float *voice, const int32_cuda rows, const int32_cuda cols, const int32_cuda *offset, const int32_cuda *bins_len, const float *bins, const int32_cuda bin_rows, const int32_cuda bin_cols, float *mel_energies_out, const int32_cuda mel_stride, int32_cuda htk_mode, int32_cuda Gr){
	_my_cuda_parallel_mel_vecvec(voice, rows, cols, offset, bins_len, bins, bin_rows, bin_cols, mel_energies_out, mel_stride, htk_mode, Gr);
} 

void _D_my_cuda_parallel_mel_vecvec(const double *voice, const int32_cuda rows, const int32_cuda cols, const int32_cuda *offset, const int32_cuda *bins_len, const double *bins, const int32_cuda bin_rows, const int32_cuda bin_cols, double *mel_energies_out, const int32_cuda mel_stride, int32_cuda htk_mode, int32_cuda Gr){
	_my_cuda_parallel_mel_vecvec(voice, rows, cols, offset, bins_len, bins, bin_rows, bin_cols, mel_energies_out, mel_stride, htk_mode, Gr);
} 

// functions for extracting window
void _F_my_cuda_process_gauss_val(float *val, int32_cuda len, const int32_cuda Gr, const int32_cuda Bl){
	_F_process_gauss_val(val, len, Gr, Bl);
}

void _F_my_cuda_scale_linear(float *A, int32_cuda dim, float alpha, int32_cuda Gr, int32_cuda Bl)
{
	_my_cuda_scale_linear(Gr, Bl, A, dim, alpha);
}

void _D_my_cuda_scale_linear(double *A, int32_cuda dim, double alpha, int32_cuda Gr, int32_cuda Bl)
{
	_my_cuda_scale_linear(Gr, Bl, A, dim, alpha);
}

void _F_my_cuda_addvec2(float *A, const float *x, int32_cuda dim, float alpha, dim3 Gr, dim3 Bl)
{
	_my_cuda_addvec2(Gr, Bl, A, x, dim, alpha);
}

void _D_my_cuda_addvec2(double *A, const double *x, int32_cuda dim, double alpha, dim3 Gr, dim3 Bl)
{
	_my_cuda_addvec2(Gr, Bl, A, x, dim, alpha);
}

void _F_my_cuda_addvec3(float *A, int32_cuda numA, const float *x, MatrixDim d, float alpha, dim3 Gr, dim3 Bl)
{
	_my_cuda_addvec3(Gr, Bl, A, numA, x, d, alpha);
}

void _D_my_cuda_addvec3(double *A, int32_cuda numA, const double *x, MatrixDim d, double alpha, dim3 Gr, dim3 Bl)
{
	_my_cuda_addvec3(Gr, Bl, A, numA, x, d, alpha);
}

void _F_my_cuda_MatVecVec(const float *A, const float *B, MatrixDim dA, MatrixDim dB, float *x, int32_cuda Gr, int32_cuda Bl)
{
	_my_cuda_MatVecVec(A, B, dA, dB, x, Gr, Bl);
}

void _D_my_cuda_MatVecVec(const double *A, const double *B, MatrixDim dA, MatrixDim dB, double *x, int32_cuda Gr, int32_cuda Bl)
{
	_my_cuda_MatVecVec(A, B, dA, dB, x, Gr, Bl);
}

void _D_my_cuda_process_gauss_val(double *val, int32_cuda len, const int32_cuda Gr, const int32_cuda Bl){
	_D_process_gauss_val(val, len, Gr, Bl);
}


void _F_my_cuda_dither(float *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda matrix_stride, float dither_value, const int32_cuda Gr, const int32_cuda Bl){
	_my_cuda_dither(waveform, rows, cols, matrix_stride, dither_value, Gr, Bl);
}

void _D_my_cuda_dither(double *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda matrix_stride, double dither_value, const int32_cuda Gr, const int32_cuda Bl){
	_my_cuda_dither(waveform, rows, cols, matrix_stride, dither_value, Gr, Bl);
}

void _F_my_cuda_preemphasize(float *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda matrix_stride, const float coeff, const int32_cuda Gr, const int32_cuda Bl){
	_my_cuda_preemphasize(waveform, rows, cols, matrix_stride, coeff, Gr, Bl);
}

void _D_my_cuda_preemphasize(double *waveform,const int32_cuda rows, const int32_cuda cols, const int32_cuda matrix_stride, const double coeff, const int32_cuda Gr, const int32_cuda Bl){
	_my_cuda_preemphasize(waveform, rows, cols, matrix_stride, coeff, Gr, Bl);
}

void _F_my_cuda_wave_sum(const float *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda matrix_stride, float *wave_sum, const int32_cuda Gr, const int32_cuda Bl){
	_my_cuda_wave_sum(waveform, rows, cols, matrix_stride, wave_sum, Gr, Bl);
}

void _D_my_cuda_wave_sum(const double *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda matrix_stride, double *wave_sum, const int32_cuda Gr, const int32_cuda Bl){
	_my_cuda_wave_sum(waveform, rows, cols, matrix_stride, wave_sum, Gr, Bl);
}

void _F_my_cuda_wave_mul(float *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda stride, const float *window, const int32_cuda Gr, const int32_cuda Bl){
	_my_cuda_wave_mul(waveform, rows, cols, stride, window, Gr, Bl);
}

void _D_my_cuda_wave_mul(double *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda stride, const double *window, const int32_cuda Gr, const int32_cuda Bl){
	_my_cuda_wave_mul(waveform, rows, cols, stride, window, Gr, Bl);
}

void _F_my_cuda_LogLikelihoodsPreselect(const int32_cuda *gselect, int32_cuda gselect_rows, int32_cuda gselect_cols, const float *features, MatrixDim d_features, const float *gconsts_, int32_cuda dim_gconsts, const float *means_invcovars_, MatrixDim d_means, const float *data_sqs, const float *inv_covars_, int32_cuda spdim, float *loglikes, MatrixDim d_loglikes, dim3 Gr, dim3 Bl)
{
	_my_cuda_LogLikelihoodsPreselect(gselect, gselect_rows, gselect_cols, features, d_features, gconsts_, dim_gconsts, means_invcovars_, d_means, data_sqs, inv_covars_, spdim, loglikes, d_loglikes, Gr, Bl);
}

void _D_my_cuda_LogLikelihoodsPreselect(const int32_cuda *gselect, int32_cuda gselect_rows, int32_cuda gselect_cols, const double *features, MatrixDim d_features, const double *gconsts_, int32_cuda dim_gconsts, const double *means_invcovars_, MatrixDim d_means, const double *data_sqs, const double *inv_covars_, int32_cuda spdim, double *loglikes, MatrixDim d_loglikes, dim3 Gr, dim3 Bl)
{
	_my_cuda_LogLikelihoodsPreselect(gselect, gselect_rows, gselect_cols, features, d_features, gconsts_, dim_gconsts, means_invcovars_, d_means, data_sqs, inv_covars_, spdim, loglikes, d_loglikes, Gr, Bl);
}

void _F_my_cuda_scale_diag_numsp(float *A, int32_cuda nums, MatrixDim d, float alpha, dim3 Gr, dim3 Bl)
{
	_my_cuda_scale_diag_numsp(A, nums, d, alpha, Gr, Bl);
}

void _D_my_cuda_scale_diag_numsp(double *A, int32_cuda nums, MatrixDim d, double alpha, dim3 Gr, dim3 Bl)
{
	_my_cuda_scale_diag_numsp(A, nums, d, alpha, Gr, Bl);
}

void _F_my_cuda_MatApplySoftMax(float *data, MatrixDim d, int32_cuda Gr, int32_cuda Bl)
{
	_my_cuda_MatApplySoftMax(data, d, Gr, Bl);
}

void _D_my_cuda_MatApplySoftMax(double *data, MatrixDim d, int32_cuda Gr, int32_cuda Bl)
{
	_my_cuda_MatApplySoftMax(data, d, Gr, Bl);
}

void _F_my_cuda_AddMatColsToVec(const float *mat, MatrixDim d, float *vec, int32_cuda Gr, int32_cuda Bl)
{
	_my_cuda_AddMatColsToVec(mat, d, vec, Gr, Bl);
}

void _F_my_cuda_set_zero(float *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda matrix_stride, const int32_cuda Gr, const int32_cuda Bl){
	_my_cuda_set_zero(waveform, rows, cols, matrix_stride, Gr, Bl);
}

void _D_my_cuda_AddMatColsToVec(const double *mat, MatrixDim d, double *vec, int32_cuda Gr, int32_cuda Bl)
{
	_my_cuda_AddMatColsToVec(mat, d, vec, Gr, Bl);
}

void _D_my_cuda_set_zero(double *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda matrix_stride, const int32_cuda Gr, const int32_cuda Bl){
	_my_cuda_set_zero(waveform, rows, cols, matrix_stride, Gr, Bl);
}

void _F_my_cuda_wave_dc_offset(float *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda matrix_stride, const float *wave_sum, const int32_cuda Gr, const int32_cuda Bl){
	 _my_cuda_wave_dc_offset(waveform, rows, cols, matrix_stride, wave_sum, Gr, Bl);
}

void _D_my_cuda_wave_dc_offset(double *waveform, const int32_cuda rows, const int32_cuda cols, const int32_cuda matrix_stride, const double *wave_sum, const int32_cuda Gr, const int32_cuda Bl){
	 _my_cuda_wave_dc_offset(waveform, rows, cols, matrix_stride, wave_sum, Gr, Bl);
}

void _F_my_cuda_log_energy(const float *src, int32_cuda rows, int32_cuda cols, int32_cuda stride, float *des){
	_F_log_energy(src, rows, cols, stride, des);
}

void _D_my_cuda_log_energy(const double *src, int32_cuda rows, int32_cuda cols, int32_cuda stride, double *des){
	_D_log_energy(src, rows, cols, stride, des);
}

void _F_my_cuda_srfft(float *wave, int32_cuda rows, int32_cuda stride, float *temp_buffer){
	_my_cuda_srfft(wave, rows, stride, temp_buffer);
}

void _D_my_cuda_srfft(double *wave, int32_cuda rows, int32_cuda stride, double *temp_buffer){
	_my_cuda_srfft(wave, rows, stride, temp_buffer);
}

void _F_my_cuda_compute_power(float *waveform, int32_cuda rows, int32_cuda cols, int32_cuda stride){
	_my_cuda_compute_power(waveform, rows, cols, stride);
}

void _D_my_cuda_compute_power(double *waveform, int32_cuda rows, int32_cuda cols, int32_cuda stride){
	_my_cuda_compute_power(waveform, rows, cols, stride);
}

void _F_my_cuda_set_energy(float *des, const int32_cuda rows, const int32_cuda stride, const float energy_floor, const float log_energy_floor, const float *src){
	_my_cuda_set_energy(des, rows, stride, energy_floor, log_energy_floor, src);
}

void _D_my_cuda_set_energy(double *des, const int32_cuda rows, const int32_cuda stride, const double energy_floor, const double log_energy_floor, const double *src){
	_my_cuda_set_energy(des, rows, stride, energy_floor, log_energy_floor, src);
}
